#include <iostream>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(const float *A, const float *B, float *C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    const int N = 2048; // vector number
    const int size = N * sizeof(float); // total size of vectors

    // allocate host memory
    float *h_A = new float[N];
    float *h_B = new float[N];
    float *h_C = new float[N];

    // initialize host data
    for (int i = 0; i < N; i++) {
        h_A[i] = i * 1.0f;
        h_B[i] = i * 2.0f;
    }

    // allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // copy data from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // launch kernel
    int threadsPerBlock = 1024;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // copy result from device to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < 10; i++) {
        std::cout << "C[" << i << "] = " << h_C[i] << std::endl;
    }

    // Clean up
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
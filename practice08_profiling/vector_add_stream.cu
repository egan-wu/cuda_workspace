#include <iostream>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(const float *A, const float *B, float *C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    const int N = 1 << 22; // vector number
    const int chunk_size = N / 2;
    const int size = chunk_size * sizeof(float); // total size of vectors
    const int total_byte = N * sizeof(float);

    // allocate host memory
    float *h_A = new float[N];
    float *h_B = new float[N];
    float *h_C = new float[N];

    // initialize host data
    for (int i = 0; i < N; i++) {
        h_A[i] = i * 1.0f;
        h_B[i] = i * 2.0f;
    }

    // allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, total_byte);
    hipMalloc(&d_B, total_byte);
    hipMalloc(&d_C, total_byte);

    // create cuda-stream
    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    // create CUDA Events
    hipEvent_t start_event, stop_event;
    hipEvent_t copy_h2d_start_event, copy_h2d_stop_event;
    hipEvent_t kernel_start_event, kernel_stop_event;
    hipEvent_t copy_d2h_start_event, copy_d2h_stop_event;

    hipEventCreate(&start_event);
    hipEventCreate(&stop_event);
    hipEventCreate(&copy_h2d_start_event);
    hipEventCreate(&copy_h2d_stop_event);
    hipEventCreate(&kernel_start_event);
    hipEventCreate(&kernel_stop_event);
    hipEventCreate(&copy_d2h_start_event);
    hipEventCreate(&copy_d2h_stop_event);

    float total_time = 0.0f;
    float h2d_time = 0.0f;
    float kernel_time = 0.0f;
    float d2h_time = 0.0f;

    // define kernel
    int threadsPerBlock = 128;
    int blocksPerGrid = (chunk_size + threadsPerBlock - 1) / threadsPerBlock;

    hipEventRecord(start_event, 0);
    {
        hipEventRecord(copy_h2d_start_event, 0);
            {
                // copy data from host to device
                // -- stream1 memcpy
                hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, stream1);
                hipMemcpyAsync(d_B, h_B, size, hipMemcpyHostToDevice, stream1);

                // -- stream2 memcpy
                hipMemcpyAsync(d_A + chunk_size, h_A + chunk_size, size, hipMemcpyHostToDevice, stream2);
                hipMemcpyAsync(d_B + chunk_size, h_B + chunk_size, size, hipMemcpyHostToDevice, stream2);
            }
            hipEventRecord(copy_h2d_stop_event, 0);
            hipEventSynchronize(copy_h2d_stop_event);

            hipEventRecord(kernel_start_event, 0);
            {
                // launch kernel
                // -- stream1 kernel
                vectorAdd<<<blocksPerGrid, threadsPerBlock, 0, stream1>>>(d_A, d_B, d_C, chunk_size);
                
                // -- stream2 kernel
                vectorAdd<<<blocksPerGrid, threadsPerBlock, 0, stream2>>>(d_A + chunk_size, d_B + chunk_size, d_C + chunk_size, chunk_size);
            }
            hipEventRecord(kernel_stop_event, 0);
            hipEventSynchronize(kernel_stop_event);

            hipEventRecord(copy_d2h_start_event, 0);
            {
                // copy result from device to host
                // -- stream1 memcpy
                hipMemcpyAsync(h_C, d_C, size, hipMemcpyDeviceToHost, stream1);
                
                // -- stream2 memcpy
                hipMemcpyAsync(h_C + chunk_size, d_C + chunk_size, size, hipMemcpyDeviceToHost, stream2);
            }
            hipEventRecord(copy_d2h_stop_event, 0);
            hipEventSynchronize(copy_d2h_stop_event);
    }
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);

    // stream synchronize
    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);

    hipEventElapsedTime(&total_time, start_event, stop_event);
    hipEventElapsedTime(&h2d_time, copy_h2d_start_event, copy_h2d_stop_event);
    hipEventElapsedTime(&kernel_time, kernel_start_event, kernel_stop_event);
    hipEventElapsedTime(&d2h_time, copy_d2h_start_event, copy_d2h_stop_event);

    for (int i = 0; i < 10; i++) {
        std::cout << "C[" << i << "] = " << h_C[i] << std::endl;
    }

    std::cout << "\n--- Configuration ---" << std::endl;
    std::cout << "chunk size: " << chunk_size << std::endl;
    std::cout << "blocks per grid: " << blocksPerGrid << std::endl;
    std::cout << "thread per block: " << threadsPerBlock << std::endl;
    std::cout << "\n--- Performance Report ---" << std::endl;
    std::cout << "Total execution time: " << total_time << " ms" << std::endl;
    std::cout << "Host to Device copy time: " << h2d_time << " ms" << std::endl;
    std::cout << "Kernel execution time: " << kernel_time << " ms" << std::endl;
    std::cout << "Device to Host copy time: " << d2h_time << " ms" << std::endl;

    // Clean up
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    hipEventDestroy(start_event);
    hipEventDestroy(stop_event);
    hipEventDestroy(copy_h2d_start_event);
    hipEventDestroy(copy_h2d_stop_event);
    hipEventDestroy(kernel_start_event);
    hipEventDestroy(kernel_stop_event);
    hipEventDestroy(copy_d2h_start_event);
    hipEventDestroy(copy_d2h_stop_event);

    return 0;
}